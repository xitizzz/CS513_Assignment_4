
#include <hip/hip_runtime.h>
#include <stdlib.h> //for rand
#include <iostream> // for cout
#include <vector>  //for vector
using std::cout;

int DIM_LIM = 100;
int MAT_COUNT = 10;

int SEED = 10; //seed for rand

//generate random sized matrices, add pointer to each, to vector
//template <typename T> //handle multiple types
class matrix {
public:
    int row; //number of rows, y
    int col; //number of columns, x
//    std::vector<int> data;
    double* data;

    matrix(int columns, int rows) :
        col(columns), row(rows),
        data(new double[col * row])
        {}

    double& getdata(int x, int y){
       return data[y * col + x]; //vertical position * row length + pos in row
    };
};


int main(){

    srand(SEED); //init random gen

    matrix* mat[MAT_COUNT]; //pointer to pointer to int
    int dim[MAT_COUNT + 1]; //stores matrix sizes
    dim[0] = rand()%DIM_LIM + 1;//random between 1 and limit


    for(int z = 0; z < MAT_COUNT; z++){

        //each matrix shares a dimension with the previous
        dim[z+1] = rand()%DIM_LIM + 1;//random between 1 and limit
        int dimx = dim[z];
        int dimy = dim[z+1];

        mat[z] = new matrix(dimx,dimy); //dimx columns, dimy rows
        for(int x = 0; x<dimx; x++){
            for(int y = 0; y<dimy; y++){
                mat[z]->getdata(x,y) = 5;
            }
        }
    }
    
    for(int z = 0; z < MAT_COUNT; z++){
        int dimxn = mat[z]->col;
        int dimyn = mat[z]->row;
        std::cout << dimxn <<" ";
        std::cout << dimyn <<" ";
        std::cout << "\n";
        for(int y = 0; y<dimyn; y++){
            for(int x = 0; x<dimxn; x++){
                cout << mat[z]->getdata(x,y) << " ";
            }
            cout << "\n";
        }
        cout << "\n";
    }

};
